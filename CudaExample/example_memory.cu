#include "example_memory.cuh"

void checkDeviceMemory(void)
{
	size_t free, total;
	hipMemGetInfo(&free, &total);
	printf("Device memory (free/total) = %lld/%lld bytes\n", free, total);
}

int mainMemory(void)
{
	int* dDataPtr;
	hipError_t errorCode;

	checkDeviceMemory();
	errorCode = hipMalloc(&dDataPtr, sizeof(int) * 1024 * 1024);
	printf("hipMalloc - %s\n", hipGetErrorName(errorCode));
	checkDeviceMemory();

	errorCode = hipMemset(dDataPtr, 0, sizeof(int) * 1024 * 1024);
	printf("hipMemset - %s\n", hipGetErrorName(errorCode));

	errorCode = hipFree(dDataPtr);
	printf("hipFree - %s\n", hipGetErrorName(errorCode));
	checkDeviceMemory();

	return 0;
}