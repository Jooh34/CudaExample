#include "hip/hip_runtime.h"
#include "matrix_mul_shared.cuh"

// Matrix size
#define SIZE_M (32)
#define SIZE_N (32)
#define SIZE_K (32*4)

#define DATA_TYPE int

__global__ void MatmulWoShared(DATA_TYPE* matA, DATA_TYPE* matB, DATA_TYPE* matC, int M, int N, int K)
{
	unsigned int col = threadIdx.y;
	unsigned int row = threadIdx.x;

	DATA_TYPE result = 0;
	if (row < M && col < N) {
		for (int k = 0; k < K; k++) {
			result += (matA[row * K + k] * matB[k * N + col]);
		}
	}
	matC[row * N + col] = result;
}

__global__ void MatmulShared(DATA_TYPE* matA, DATA_TYPE* matB, DATA_TYPE* matC, int M, int N, int K)
{
	unsigned int col = threadIdx.y;
	unsigned int row = threadIdx.x;
	
	__shared__ float sA[SIZE_M][SIZE_K]; // 32*128*4 bytes = 16KB
	__shared__ float sB[SIZE_K][SIZE_N]; // 32*128*4 bytes = 16KB

	if (row == 0) {
		for (int k = 0; k < K; k++) {
			sB[k][col] = matB[col + k * N];
		}
	}

	if (col == 0 ) {
		for (int k = 0; k < K; k++) {
			sA[row][k] = matA[k + row * K];
		}
	}

	__syncthreads();

	DATA_TYPE result = 0;
	if (row < M && col < N) {
		for (int k = 0; k < K; k++) {
			result += (sA[row][k] * sB[k][col]);
		}
	}
	matC[row * N + col] = result;
}

int mainMatmulShared(bool bUseSharedMemory)
{
	// set matrix size
	int m, n, k;
	m = SIZE_M;
	n = SIZE_N;
	k = SIZE_K;

	printf("Size : A = (%d by %d), B = (%d by %d), C = (%d by %d)\n", m, k, k, n, m, n);

	int sizeA = m * k;
	int sizeB = k * n;
	int sizeC = m * n;

	// Make matrix
	DATA_TYPE* A = NULL, * B = NULL;
	allocNinitMem<DATA_TYPE>(&A, sizeA);
	allocNinitMem<DATA_TYPE>(&B, sizeB);

	DATA_TYPE* Ccpu = NULL, * Cgpu = NULL;
	allocNinitMem<DATA_TYPE>(&Ccpu, sizeC);
	allocNinitMem<DATA_TYPE>(&Cgpu, sizeC);

	// generate input matrices
	for (int i = 0; i < sizeA; i++) A[i] = ((rand() % 10) + ((rand() % 100) / 100.0));
	for (int i = 0; i < sizeB; i++) B[i] = ((rand() % 10) + ((rand() % 100) / 100.0));

	{
		SCOPED_TIMER("CPU matmul");
		// CPU matmul
		for (int row = 0; row < m; row++) {
			for (int col = 0; col < n; col++) {
				int cIndex = row * n + col;
				Ccpu[cIndex] = 0;
				for (int i = 0; i < k; i++)
					Ccpu[cIndex] += (A[row * k + i] * B[i * n + col]);
			}
		}
	}

	/******************************************************************
	* Write your codes for GPU algorithm from here
	******************************************************************/
	DATA_TYPE* dA, * dB, * dC;

	// 1. Allocate device memory for dA, dB, dC
	{
		SCOPED_TIMER("CUDA malloc and memset");
		hipMalloc(&dA, sizeof(DATA_TYPE) * sizeA); hipMemset(dA, 0, sizeof(DATA_TYPE) * sizeA);
		hipMalloc(&dB, sizeof(DATA_TYPE) * sizeB); hipMemset(dB, 0, sizeof(DATA_TYPE) * sizeB);
		hipMalloc(&dC, sizeof(DATA_TYPE) * sizeC); hipMemset(dC, 0, sizeof(DATA_TYPE) * sizeC);
	}

	// 2. Send(Copy) the input matrices to GPU (A -> dB, B -> dB)
	{
		SCOPED_TIMER("CUDA copy Host -> Device");
		hipMemcpy(dA, A, sizeof(DATA_TYPE) * sizeA, hipMemcpyHostToDevice);
		hipMemcpy(dB, B, sizeof(DATA_TYPE) * sizeB, hipMemcpyHostToDevice);
	}


	// 3. Set the thread layout
	if (bUseSharedMemory)
	{
		dim3 blockDim(32, 32, 1);
		dim3 gridDim(1, 1, 1);

		printf("Grid(%d, %d), Block(%d, %d)\n", gridDim.x, gridDim.y, blockDim.x, blockDim.y);
		{
			SCOPED_TIMER("MatmulShared on GPU");
			MatmulShared << < gridDim, blockDim >> > (dA, dB, dC, m, n, k);
			hipDeviceSynchronize(); // this is synchronization for mearusing the kernel processing time
		}
	}
	else
	{
		dim3 blockDim(32, 32, 1);
		dim3 gridDim(1, 1, 1);

		printf("Grid(%d, %d), Block(%d, %d)\n", gridDim.x, gridDim.y, blockDim.x, blockDim.y);

		{
			SCOPED_TIMER("MatmulWoShared on GPU");
			MatmulWoShared << < gridDim, blockDim >> > (dA, dB, dC, m, n, k);
			hipDeviceSynchronize(); // this is synchronization for mearusing the kernel processing time
		}
	}

	//5. Get(copy) the result from GPU to host memory (dC -> Cgpu)
	{
		SCOPED_TIMER("CUDA copy Device -> Host");
		hipMemcpy(Cgpu, dC, sizeof(DATA_TYPE) * sizeC, hipMemcpyDeviceToHost);
	}

	// 6. Release device memory space (dA, dB, dC)
	hipFree(dA); hipFree(dB); hipFree(dC);


	compareMatrix<DATA_TYPE>(Ccpu, Cgpu, sizeC);

	Timer::getInstance().printRecord();

	delete A;
	delete B;
	delete Ccpu;
	delete Cgpu;

	return 0;
}