﻿#include "kernel.cuh"
#include "example_memory.cuh"
#include "vector_addition.cuh"
#include "vector_addition_large.cuh"
#include "matrix_addition_large.cuh"
#include "matrix_mul.cuh"
#include "matrix_mul_shared.cuh"
#include "matrix_mul_large_shared.cuh"
#include "matrix_mul_memacc_opt.cuh"

#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"

#include <stdio.h>

#define _1MB (1024*1024)

void deviceQuery() {
    int ngpus;
    hipGetDeviceCount(&ngpus);

    for (int i = 0; i < ngpus; i++) {
        hipDeviceProp_t devProp;

        hipGetDeviceProperties(&devProp, i);

        printf("Device %d: %s\n"
            , i, devProp.name);
        printf("\tCompute capability: %d.%d\n"
            , devProp.major, devProp.minor);
        printf("\tThe number of streaming multiprocessors: %d\n"
            , devProp.multiProcessorCount);
        printf("\tThe number of CUDA cores: %d\n"
            , _ConvertSMVer2Cores(devProp.major, devProp.minor)
            * devProp.multiProcessorCount);
        printf("\tGlobal memory size: %.2f MB\n"
            , (float)devProp.totalGlobalMem / _1MB);
    }
}

int main()
{
    deviceQuery();
    //mainMatmul(BlockType::B2D_G2D);
	//mainMatmulShared(false);
    mainMatmulLargeShared(true, true);
    //mainMatmulMemaccOpt(true);
	return 0;
}