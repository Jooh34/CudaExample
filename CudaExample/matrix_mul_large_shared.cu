#include "hip/hip_runtime.h"
#include "matrix_mul_large_shared.cuh"

// Matrix size
#define SIZE_M (512*2)
#define SIZE_N (512*4)
#define SIZE_K (512*2)

#define DATA_TYPE int

#define BLOCK_X 8
#define BLOCK_Y 8
#define BLOCK_K 8

__global__ void MatMulLarge(DATA_TYPE* matA, DATA_TYPE* matB, DATA_TYPE* matC, int M, int N, int K, int gridnumK)
{
	unsigned int row = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int col = blockDim.y * blockIdx.y + threadIdx.y;

	__shared__ DATA_TYPE sA[BLOCK_Y][BLOCK_K];
	__shared__ DATA_TYPE sB[BLOCK_K][BLOCK_X];
	//__shared__ DATA_TYPE sC[BLOCK_Y][BLOCK_X]; // shared memory for save 

	/*for (int x = 0; x < BLOCK_X; x++) {
		for (int y = 0; y < BLOCK_Y; y++) {
			sC[y][x] = 0;
		}
	}*/

	DATA_TYPE result = 0;
	for (int k_gridcol = 0; k_gridcol < gridnumK; k_gridcol++) {
		for (int k_gridrow = 0; k_gridrow < gridnumK; k_gridrow++) {
			// shared memory sB initialization
			if (threadIdx.x == 0) {
				for (int k = 0; k < BLOCK_K; k++) {
					int k_index = k_gridrow * BLOCK_K + k;
					if (k_index < K) {
						sB[k][threadIdx.y] = matB[col + k_index * N];
					}
				}
			}

			// shared memory sA initialization
			if (threadIdx.y == 0) {
				for (int k = 0; k < BLOCK_K; k++) {
					int k_index = k_gridcol * BLOCK_K + k;
					if (k_index < K) {
						sA[threadIdx.x][k] = matA[k_index + row * K];
					}
				}
			}

			__syncthreads();

			for (int k = 0; k < BLOCK_K; k++) {
				result += (sA[threadIdx.x][k] * sB[k][threadIdx.y]);
			}
		}
	}

	matC[row * N + col] = result;
}

int mainMatmulLargeShared()
{
	// set matrix size
	int m, n, k;
	m = SIZE_M;
	n = SIZE_N;
	k = SIZE_K;

	printf("Size : A = (%d by %d), B = (%d by %d), C = (%d by %d)\n", m, k, k, n, m, n);

	int sizeA = m * k;
	int sizeB = k * n;
	int sizeC = m * n;

	// Make matrix
	DATA_TYPE* A = NULL, * B = NULL;
	allocNinitMem<DATA_TYPE>(&A, sizeA);
	allocNinitMem<DATA_TYPE>(&B, sizeB);

	DATA_TYPE* Ccpu = NULL, * Cgpu = NULL;
	allocNinitMem<DATA_TYPE>(&Ccpu, sizeC);
	allocNinitMem<DATA_TYPE>(&Cgpu, sizeC);

	// generate input matrices
	for (int i = 0; i < sizeA; i++) A[i] = ((rand() % 10) + ((rand() % 100) / 100.0));
	for (int i = 0; i < sizeB; i++) B[i] = ((rand() % 10) + ((rand() % 100) / 100.0));

	{
		SCOPED_TIMER("CPU matmul");
		// CPU matmul
		for (int row = 0; row < m; row++) {
			for (int col = 0; col < n; col++) {
				int cIndex = row * n + col;
				Ccpu[cIndex] = 0;
				for (int i = 0; i < k; i++)
					Ccpu[cIndex] += (A[row * k + i] * B[i * n + col]);
			}
		}
	}

	/******************************************************************
	* Write your codes for GPU algorithm from here
	******************************************************************/
	DATA_TYPE* dA, * dB, * dC;

	// 1. Allocate device memory for dA, dB, dC
	{
		SCOPED_TIMER("CUDA malloc and memset");
		hipMalloc(&dA, sizeof(DATA_TYPE) * sizeA); hipMemset(dA, 0, sizeof(DATA_TYPE) * sizeA);
		hipMalloc(&dB, sizeof(DATA_TYPE) * sizeB); hipMemset(dB, 0, sizeof(DATA_TYPE) * sizeB);
		hipMalloc(&dC, sizeof(DATA_TYPE) * sizeC); hipMemset(dC, 0, sizeof(DATA_TYPE) * sizeC);
	}

	// 2. Send(Copy) the input matrices to GPU (A -> dB, B -> dB)
	{
		SCOPED_TIMER("CUDA copy Host -> Device");
		hipMemcpy(dA, A, sizeof(DATA_TYPE) * sizeA, hipMemcpyHostToDevice);
		hipMemcpy(dB, B, sizeof(DATA_TYPE) * sizeB, hipMemcpyHostToDevice);
	}


	// 3. Set the thread layout
	{
		dim3 blockDim(BLOCK_X, BLOCK_Y, 1);
		dim3 gridDim(ceil(float(m) / blockDim.x), ceil(float(n) / blockDim.y), 1);
		int gridnumK = ceilf((float)k / BLOCK_K);

		printf("Grid(%d, %d), Block(%d, %d)\n", gridDim.x, gridDim.y, blockDim.x, blockDim.y);

		// 4. Kernel call
		{
			SCOPED_TIMER("Matmul on GPU");
			MatMulLarge << < gridDim, blockDim >> > (dA, dB, dC, m, n, k, gridnumK);
			hipDeviceSynchronize(); // this is synchronization for mearusing the kernel processing time
		}
	}

	//5. Get(copy) the result from GPU to host memory (dC -> Cgpu)
	{
		SCOPED_TIMER("CUDA copy Device -> Host");
		hipMemcpy(Cgpu, dC, sizeof(DATA_TYPE) * sizeC, hipMemcpyDeviceToHost);
	}

	// 6. Release device memory space (dA, dB, dC)
	hipFree(dA); hipFree(dB); hipFree(dC);


	compareMatrix<DATA_TYPE>(Ccpu, Cgpu, sizeC);

	Timer::getInstance().printRecord();

	delete A;
	delete B;
	delete Ccpu;
	delete Cgpu;

	return 0;
}
